FFT::StatusType FFT::computeBackwardBatched(ComplexTypePtr src, RealTypePtr dst)
    {
        StatusType s = hipfftExecC2R(backward_handle_, reinterpret_cast<hipfftComplex*>(src), dst);
        dim3 gridSize((getBatchSize() * getForwardDataSize()) / thr_max_ + 1, 1, 1);
        dim3 blockSize(thr_max_, 1, 1);
        float val = getForwardDataSize();
        kernelDivide <<< gridSize, blockSize >>> (dst, val, getBatchSize() * getForwardDataSize());
        return s;
    }

    FFT::StatusType FFT::computeBackwardBatched(ComplexVector& src, Vector& dst)
    {
        return computeBackwardBatched(src.getData(), dst.getData());
    }
